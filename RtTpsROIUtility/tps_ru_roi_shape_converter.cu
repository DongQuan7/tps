#include "hip/hip_runtime.h"
#include "RtTpsROIUtility/tps_ru_defs.h"
#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"
#include "hip/hip_runtime_api.h"

#include "hip/hip_math_constants.h"
#include ""
#include "hip/device_functions.h"
#include ""

#include "tps_ru_common.h"

TPS_BEGIN_NAMESPACE

__device__ float2 project(contour* ctr, float2 axis)
{
    float dot_proj = dot(axis, ctr->points[0]);
    float2 proj = make_float2(dot_proj, dot_proj);
    for (int i = 1; i < ctr->count; ++i)
    {
        dot_proj = dot(axis, ctr->points[i]);
        proj.x = min(proj.x, dot_proj);
        proj.y = max(proj.y, dot_proj);
    }
    return proj;
}

__device__ bool has_overlap(contour* ctr0, contour* ctr1)
{
    if (ctr0->count < 3 || ctr1->count < 3) return false;

    float4 bdry0 = make_float4(
        ctr0->points[0].x, ctr0->points[0].x, 
        ctr0->points[0].y, ctr0->points[0].y);
    for (int i = 1; i < ctr0->count; ++i)
    {
        bdry0.x = min(bdry0.x, ctr0->points[i].x);
        bdry0.y = max(bdry0.y, ctr0->points[i].x);
        bdry0.z = min(bdry0.z, ctr0->points[i].y);
        bdry0.w = max(bdry0.w, ctr0->points[i].y);
    }

    float4 bdry1 = make_float4(
        ctr1->points[0].x, ctr1->points[0].x, 
        ctr1->points[0].y, ctr1->points[0].y);
    for (int i = 1; i < ctr1->count; ++i)
    {
        bdry1.x = min(bdry1.x, ctr1->points[i].x);
        bdry1.y = max(bdry1.y, ctr1->points[i].x);
        bdry1.z = min(bdry1.z, ctr1->points[i].y);
        bdry1.w = max(bdry1.w, ctr1->points[i].y);
    }

    if (bdry0.x >= bdry1.y || bdry1.x >= bdry0.y || 
        bdry0.z >= bdry1.w || bdry1.z >= bdry0.w)
    {
        return false;
    }
    return true;

    //����ֻ�����жϣ�SAT����̫��

    //using separating axis theorem to detect overlapping
    //float2 axis, proj0, proj1;
    //for (int i = 0; i < ctr0->count; ++i)
    //{
    //    axis = ctr0->points[(i+1) % ctr0->count] - ctr0->points[i];
    //    if (length(axis) <= 1e-6) continue;
    //    //project contour 0
    //    proj0 = project(ctr0, axis);
    //    //project contour 1
    //    proj1 = project(ctr1, axis);

    //    if (proj0.x >= proj1.y || proj1.x >= proj0.y)
    //    {
    //        return false;
    //    }
    //}

    //for (int i = 0; i < ctr1->count; ++i)
    //{
    //    axis = ctr1->points[(i+1) % ctr1->count] - ctr1->points[i];
    //    if (length(axis) <= 1e-6) continue;
    //    //project contour 0
    //    proj0 = project(ctr0, axis);
    //    //project contour 1
    //    proj1 = project(ctr1, axis);

    //    if (proj0.x >= proj1.y || proj1.x >= proj0.y)
    //    {
    //        return false;
    //    }
    //}
    //return true;
}

__device__ bool has_overlap(contour* pc, int count, contour* ctr)
{
    for (int i = 0; i < count; ++i)
    {
        if (has_overlap(pc+i, ctr))
        {
            return true;
        }
    }
    return false;
}

__device__ bool is_on_segment(float2 p0, float2 p1, float2 p)
{
    float2 v_seg = p1 - p0;
    float2 v = p - p0;

    float cross_proj = v.y * v_seg.x - v.x * v_seg.y;
    if (abs(cross_proj) > 1e-6) return false;

    float dot_proj = dot(v, v_seg);
    if (dot_proj < 0) return false;

    float squr_len = v_seg.x * v_seg.x + v_seg.y * v_seg.y;
    if (dot_proj > squr_len) return false;

    return true;
}

__device__ char calc_sdf_label(contour* contours, int count, float2 point)
{
    int cn = 0, vc;
    float lambda;
    float2 p0, p1;
    char sdf_label = 0;
    for (int i = 0; i < count; ++i)
    {
        vc = contours[i].count;
        if (vc < 3) continue;

        int n = 0;
        p0 = contours[i].points[vc-1];
        for (int j = 0; j < vc; ++j)
        {
            p1 = contours[i].points[j];
            if (length(p1 - p0) <= 1e-6) continue;

            if (is_on_segment(p0, p1, point))
            {
                //on the contour
                sdf_label |= (contours[i].hat_flag<<2);
                sdf_label |= 0x01;
                return sdf_label;
            }

            if ((p0.y <= point.y && p1.y > point.y) || 
                (p1.y <= point.y && p0.y > point.y))
            {
                lambda = __fdividef(__fsub_rn(point.y, p0.y), __fsub_rn(p1.y, p0.y));
                if (__fadd_rn(p0.x, __fmul_rn(lambda, __fsub_rn(p1.x, p0.x))) > point.x)
                    ++n;
            }
            p0 = p1;
        }
        //if in the contour, dump the hat flag
        if (n % 2 != 0)
        {
            sdf_label |= (contours[i].hat_flag<<2);
        }
        cn += n;
    }
    if (cn % 2 != 0) sdf_label |= 0x02;
    return sdf_label;
}

__device__ float calc_dist_to_segment(float2 start, float2 end, float2 point)
{
    float2 v0 = end - start;
    float seg_len = length(v0);
    if (seg_len <= 1e-6)
    {
        return length(point - start);
    }
    float2 norm = normalize(v0);
    float2 v1 = point - start;
    float ratio = dot(v1, norm) / seg_len;
    if (ratio <= 0)
    {
        return length(v1);
    }
    else if (ratio >= 1)
    {
        return length(point - end);
    }
    else
    {
        return length(ratio * v0 - v1);
    }
}

__device__ bool is_neighbor_all_same(int3 p, char* curr, int slice_pitch, int pitch, int3 dim)
{
    bool curr_is_out = (*curr & 0x03) == 0;
    for (int k = -NEIGHBOR_SHIFT; k <= NEIGHBOR_SHIFT; ++k)
    {
        if (p.z + k < 0 || p.z + k > dim.z - 1)
        {
            if (!curr_is_out) return false;
            continue;
        }
        for (int j = -NEIGHBOR_SHIFT; j <= NEIGHBOR_SHIFT; ++j)
        {
            if (p.y + j < 0 || p.y + j > dim.y - 1)
            {
                if (!curr_is_out) return false;
                continue;
            }
            for (int i = -NEIGHBOR_SHIFT; i <= NEIGHBOR_SHIFT; ++i)
            {
                if (p.x + i < 0 || p.x + i > dim.x - 1)
                {
                    if (!curr_is_out) return false;
                    continue;
                }
                char* ch = curr + k * slice_pitch + j * pitch + i;
                if (((*ch & 0x03) == 0) ^ curr_is_out) return false;
            }
        }
    }
    return true;
}

__device__ bool is_neighbor_dirty(contour* contours, 
    int2* pci, int page_count, int curr_page)
{
    int shift = NEIGHBOR_SHIFT + 1;
    bool any_neighbor_dirty = false;
    for (int i = -shift; i <= shift; ++i)
    {
        int idx = curr_page + i;
        if (idx < 0 || idx >= page_count || pci[idx].y == 0) continue;
        if (contours[pci[idx].x].is_dirty)
        {
            any_neighbor_dirty = true;
            break;
        }
    }
    return any_neighbor_dirty;
}

__device__ float calc_min_dist(contour* contours, int count, float2 point)
{
    int vc;
    float2 p0, p1;
    float min_dist = HIP_MAX_NORMAL_F;
    for (int i = 0; i < count; ++i)
    {
        vc = contours[i].count;
        if (vc < 3) continue;

        p0 = contours[i].points[vc-1];
        for (int j = 0; j < vc; ++j)
        {
            p1 = contours[i].points[j];
            min_dist = min(min_dist, calc_dist_to_segment(p0, p1, point));
            p0 = p1;
        }
    }
    return min_dist;
}


__global__ void ContoursOverlappingLabelling(contour* contours, 
    int2* pci, int page_count, bool only_dirty)
{
    int ctr_count = pci[page_count-1].x + pci[page_count-1].y;
    int ctr_idx = blockIdx.x * blockDim.x * blockDim.y + 
        threadIdx.y * blockDim.x + threadIdx.x;
    if (ctr_idx >= ctr_count) return;

    int curr_page = -1;
    contour* ctr = contours + ctr_idx;
    for (int i = 0; i < page_count; ++i)
    {
        if (ctr_idx < pci[i].x + pci[i].y)
        {
            curr_page = i;
            break;
        }
    }
    if (curr_page == -1) return;

    ctr->hat_flag = 0;

    //if only dirty slices, check neighbor slice dirty flag
    if (only_dirty && !is_neighbor_dirty(contours, pci, page_count, curr_page))
    {
        return;
    }

    //pre page
    if (curr_page == 0 || 
        !has_overlap(contours + pci[curr_page-1].x, pci[curr_page-1].y, ctr))
    {
        ctr->hat_flag |= 0x01;
    }
    //post page
    if (curr_page == page_count-1 || 
        !has_overlap(contours + pci[curr_page+1].x, pci[curr_page+1].y, ctr))
    {
        ctr->hat_flag |= 0x02;
    }
}

__global__ void SDFLabelling(contour* contours, int2* pci, 
    int3 dim, float3 spacing, hipPitchedPtr label, bool only_dirty)
{
    int3 point = make_int3(
        blockIdx.x * blockDim.x + threadIdx.x, 
        blockIdx.y * blockDim.y + threadIdx.y,
        blockIdx.z);
    if (point.x >= dim.x || point.y >= dim.y)
        return;

    if (only_dirty && !is_neighbor_dirty(contours, pci, dim.z, point.z))
    {
        return;
    }

    size_t pitch = label.pitch;
    size_t slice_pitch = pitch * dim.y;
    char* tgt = (char*)label.ptr + point.z * slice_pitch + 
        point.y * pitch + point.x;

    int start = pci[point.z].x;
    int count = pci[point.z].y;
    if (count == 0)
    {
        *tgt = 0;
        return;
    }

    float2 fpt = make_float2(point.x * spacing.x, point.y * spacing.y);
    contour* seg_start = contours + start;
    *tgt = calc_sdf_label(seg_start, count, fpt);
}

__device__ float calc_sdf(contour* contours, int2* pci, int3 dim, 
    float3 spacing, hipPitchedPtr label, int3 point)
{
    float default_sdf = spacing.z * 0.5f;
    int count = pci[point.z].y;
    if (count == 0)
    {
        return -default_sdf;
    }

    size_t lbl_pitch = label.pitch;
    size_t lbl_slice_pitch = lbl_pitch * dim.y;
    char* curr_lbl = (char*)label.ptr + point.z * lbl_slice_pitch + 
        point.y * lbl_pitch + point.x;

    //if on contour, sdf is 0
    if ((*curr_lbl & 0x01) != 0)
    {
        return 0.f;
    }

    int sign = 0;
    bool limited_to_half_spacing = false;
    //if out of contour
    if ((*curr_lbl & 0x03) == 0)
    {
        char* down_lbl = point.z == 0 ? 0 : (curr_lbl - lbl_slice_pitch);
        char* up_lbl = point.z == dim.z - 1 ? 0 : (curr_lbl + lbl_slice_pitch);
        sign = -1;
        //if up is in down-hatted contour or down is in up-hatted contour, set limit flag
        if (((*up_lbl & 0x04) != 0) || ((*down_lbl & 0x08) != 0)) 
        {
            limited_to_half_spacing = true;
        }
    }
    //if in a contour
    else
    {
        sign = 1;
        // if current is in hatted contour, set limit flag
        if ((*curr_lbl & 0x0c) != 0)
        {
            limited_to_half_spacing = true;
        }
    }
    if (limited_to_half_spacing)
    {
        return sign * default_sdf;
    }

    //check 5*5*5 cube, if all in or all out
//     if (is_neighbor_all_same(point, curr_lbl, lbl_slice_pitch, lbl_pitch, dim))
//     {
//         return sign * length(spacing) * 0.5f;
//     }

    float2 fp = make_float2(point.x * spacing.x, point.y * spacing.y);
    contour* ctr_start = contours + pci[point.z].x;
    float min_dist = calc_min_dist(ctr_start, count, fp);
    return min_dist * sign;
}

__device__ float calc_sdf_without_hat(contour* contours, int count, float2 point)
{
    int cn = 0;
    float lambda;
    float2 p0, p1;
    float min_dist = HIP_MAX_NORMAL_F;
    for (int i = 0; i < count; ++i)
    {
        int vc = contours[i].count;
        if (vc < 3) continue;

        p0 = contours[i].points[vc-1];
        for (int j = 0; j < vc; ++j)
        {
            p1 = contours[i].points[j];
            if (length(p1 - p0) <= 1e-6) continue;

            if ((p0.y <= point.y && p1.y > point.y) || 
                (p1.y <= point.y && p0.y > point.y))
            {
                lambda = __fdividef(__fsub_rn(point.y, p0.y), __fsub_rn(p1.y, p0.y));
                if (__fadd_rn(p0.x, __fmul_rn(lambda, __fsub_rn(p1.x, p0.x))) > point.x)
                    ++cn;
            }
            min_dist = min(min_dist, calc_dist_to_segment(p0, p1, point));
            p0 = p1;
        }
    }
    int sign = (cn % 2 == 0) ? -1: 1;
    return sign * min_dist;
}

__global__ void CalculateSDF(contour* contours, int2* pci, int3 dim, 
    float3 spacing, hipPitchedPtr label, hipPitchedPtr sdf)
{
    int3 p = make_int3(
        blockIdx.x * blockDim.x + threadIdx.x, 
        blockIdx.y * blockDim.y + threadIdx.y,
        blockIdx.z);
    if (p.x >= dim.x || p.y >= dim.y) return;

    size_t sdf_pitch = sdf.pitch;
    size_t sdf_slice_pitch = sdf_pitch * dim.y;
    float* tgt = (float*)((char*)sdf.ptr + p.z * sdf_slice_pitch + 
        p.y * sdf_pitch) + p.x;

    *tgt = calc_sdf(contours, pci, dim, spacing, label, p);
}

__global__ void CalculateSDF_SingleSlice(contour* contours, int2* pci, int3 dim, 
    float3 spacing, hipPitchedPtr label, int slice_num, float* plane_sdf)
{
    int3 p = make_int3(
        blockIdx.x * blockDim.x + threadIdx.x, 
        blockIdx.y * blockDim.y + threadIdx.y,
        slice_num);
    if (p.x >= dim.x || p.y >= dim.y) return;

    plane_sdf[p.y * dim.x + p.x] = calc_sdf(contours, pci, dim, spacing, label, p);
}

__global__ void CalculateSDFWithoutHat_SingleSlice(contour* contours, 
    int count, int2 dim, float3 spacing, float* plane_sdf)
{
    int2 p = make_int2(
        blockIdx.x * blockDim.x + threadIdx.x, 
        blockIdx.y * blockDim.y + threadIdx.y);
    if (p.x >= dim.x || p.y >= dim.y) return;

    if (count == 0)
    {
        plane_sdf[p.y * dim.x + p.x] = -spacing.z / 2;
        return;
    }

    float2 fp = make_float2(p.x * spacing.x, p.y * spacing.y);
    plane_sdf[p.y * dim.x + p.x] = calc_sdf_without_hat(contours, count, fp);
}

extern "C"
void Launch_ContoursOverlappingLabelling(dim3 grid, dim3 threads, contour* contours, 
    int2* page_contours_info, int page_count, bool only_dirty)
{
    ContoursOverlappingLabelling<<<grid, threads>>>(contours, page_contours_info, page_count, only_dirty);
    getLastCudaError("ContoursOverlappingLabelling failed");
}

extern "C"
void Launch_SDFLabelling(dim3 grid, dim3 threads, contour* contours, 
    int2* pci, int3 dim, float3 spacing, hipPitchedPtr label, bool only_dirty)
{
    SDFLabelling<<<grid, threads>>>(contours, pci, dim, spacing, label, only_dirty);
    getLastCudaError("SDFLabelling failed");
}

extern "C"
void Launch_CalculateSDF(dim3 grid, dim3 threads, contour* contours, 
    int2* pci, int3 dim, float3 spacing, hipPitchedPtr label, hipPitchedPtr sdf)
{
    CalculateSDF<<<grid, threads>>>(contours, pci, dim, spacing, label, sdf);
    getLastCudaError("CalculateSDF failed");
}

extern "C"
void Launch_CalculateSDF_SingleSlice(dim3 grid, dim3 threads, contour* contours, 
    int2* pci, int3 dim, float3 spacing, hipPitchedPtr label, int slice_num, float* plane_sdf)
{
    CalculateSDF_SingleSlice<<<grid, threads>>>(
        contours, pci, dim, spacing, label, slice_num, plane_sdf);
    getLastCudaError("Launch_CalculateSDF_SinglePlane failed");
}

extern "C"
void Launch_CalculateSDFWithoutHat_SingleSlice(dim3 grid, dim3 threads, contour* contours, 
    int count, int2 dim, float3 spacing, float* plane_sdf)
{
    CalculateSDFWithoutHat_SingleSlice<<<grid, threads>>>(
        contours, count, dim, spacing, plane_sdf);
    getLastCudaError("CalculateSDFWithoutHat_SingleSlice failed");
}

__global__ void MarchingSquareLabelling(hipTextureObject_t texObj, 
    double3 start, double3 vx, double3 vy, float iso_level, MS_MAP ms_map)
{
    int2 square_idx;
    square_idx.x = blockIdx.x * blockDim.x + threadIdx.x;
    square_idx.y = blockIdx.y * blockDim.y + threadIdx.y;
    if(square_idx.x >= ms_map.map_size.x || 
       square_idx.y >= ms_map.map_size.y)
    {
        return;
    }

    double3 p0 = start + vx * square_idx.x + vy * square_idx.y;
    double3 corners[4] = { p0, p0 + vx, p0 + vx + vy, p0 + vy };
    float sd_square[4];
    for(int i = 0; i < 4; ++i)
    {
        //use double here to avoid float precision loss problem 
        sd_square[i] = tex3D<float>(
            texObj, corners[i].x, corners[i].y, corners[i].z) - iso_level;
    }

    char id = 0;
    for(int i = 0; i < 4; ++i)
    {
        if(sd_square[i] >= -1e-6) id |= (0x01<<i);
    }
    char* label = ms_map.label_map + 
        square_idx.y * ms_map.l_pitch + square_idx.x;
    *label = char(id << 0x04);
    if(id == 0 || id == 15 || id == 4 || id == 11) return;

    float3* r_v = (float3*)((char*)ms_map.row_vertex_map + 
        square_idx.y * ms_map.v_pitch) + square_idx.x;
    float3* c_v = (float3*)((char*)ms_map.column_vertex_map + 
        square_idx.y * ms_map.v_pitch) + square_idx.x;
    if(abs(sd_square[0]) <= 1e-6)
    {
        *r_v = make_float3(p0);
        *c_v = make_float3(p0);
        *label |= 0x03;
    }
    else 
    {
        if(sd_square[0] * sd_square[1] <= 1e-6)
        {
            *r_v = make_float3(p0 - sd_square[0] / (sd_square[1] - sd_square[0]) * vx);
            *label |= 0x01;
        }
        if(sd_square[0] * sd_square[3] <= 1e-6)
        {
            *c_v = make_float3(p0 - sd_square[0] / (sd_square[3] - sd_square[0]) * vy);
            *label |= 0x02;
        }
    }

    //ambiguous case
    if (id == 5 || id == 10)
    {
        float3 center = make_float3(p0 + 0.5 * vx + 0.5 * vy);
        float central_sd = tex3D<float>(texObj, center.x, center.y, center.z);
        if (central_sd >= 0)
        {
            *label |= 0x04;
        }
    }
}

extern "C"
void Launch_MarchingSquareLabelling(dim3 grid, dim3 threads, hipTextureObject_t texObj, 
    double3 start, double3 vx, double3 vy, float iso_level, MS_MAP ms_map)
{
    MarchingSquareLabelling<<<grid, threads>>>(texObj, start, vx, vy, iso_level, ms_map);
    getLastCudaError("Launch_MarchingSquareLabelling failed!");
}

TPS_END_NAMESPACE